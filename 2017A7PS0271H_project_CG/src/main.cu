#include "hip/hip_runtime.h"
/**@file main.cu */
#include "hip/hip_runtime.h"
#include ""
#include<GL/glew.h>
#include<iostream>
#include <stdio.h>
#include <GLFW/glfw3.h>
#include "Shader.h"
#include "VertexBuffer.h"
#include "IndexBuffer.h"
#include "vert_array_quick_bind.h"
#include "Global_Bind_Test.h"
#include "freecam.h"
#include <functional>
#include <fstream>
#include <sstream>
#include "artefact.h"
#include <stb_image/stb_image.h>
#include "cudaGL.h"
#include "cuda_gl_interop.h"
#include <thrust/device_vector.h>
#include "Texture.h"
#include "ray.cuh"
#include "sphere.cuh"
#include "scene.cuh"
#include <float.h>
#include "ray_tracing_camera.cuh"
#include <hiprand/hiprand_kernel.h>
#include <glm/gtc/random.hpp>

using namespace glm;
///error checking support
#define gpuCheckErrs(ans) { gpuAssert((ans), __FILE__, __LINE__); }
///call inside this function for CUDA error checking.
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

using namespace std;
///The raster camera, not really needed, but since we are using an unmodified older shader that does need this, we left it in.
freecam primary_cam;
///Wrapper function for the mouse controller
void MouseControlWrapper(GLFWwindow* window, double mouse_x, double mouse_y) {
	primary_cam.mouse_handler(window, mouse_x, mouse_y);
}
///Wrapper class for the scroll controller
void ScrollControlWrapper(GLFWwindow* window, double x_disp, double y_disp) {
	primary_cam.scroll_handler(window, x_disp, y_disp);
}
///A random state uniform from the CuRanD library is returned
#define RANDVEC3 vec3(hiprand_uniform(local_rand_state),hiprand_uniform(local_rand_state),hiprand_uniform(local_rand_state))
///@param local_rand_state A pointer the the CUDA random state
///
///Returns a point inside a unit sphere on the surface of the sphere where the ray
/// intersection takes place.
__device__ vec3 random_in_unit_sphere(hiprandState* local_rand_state) {
	vec3 p;
	do {
		p = 2.0f * RANDVEC3 - vec3(1, 1, 1);
	} while ((length(p)) * (length(p)) >= 1.0f);
	return p;
}
///@param spheres device pointer to the memory segment on the VRAM where the sphere definitions are present.
///@param dev_ptr dev_ptr device pointer to the GPU memory segment where the scene is to be set up.
///@param count The number of spheres in the scene.
__global__ void make_scene(sphere** spheres, scene** dev_ptr, int count) {
	*dev_ptr = new scene(spheres, count);
}
///@param v The incident ray direction.
///@param n The unit vector of the normal at the intersection point.
///
///Returns the direction of the reflected
/// ray.
/// The __device__ declspec means that this can only be called on the GPU.
__device__ vec3 reflect(vec3 v, vec3 n) {
	return v - 2 * dot(v, n) * n;
}
///@param uv The direction vector of the incident ray.
///@param n  The normal at the point where the ray intersects the sphere.
/// @param etai_over_etat The refractive index of the spheres material.
///
///Returns the direction of the refracted ray
/// the __device__ declspec means that this can only be called on the GPU.
__device__ vec3 refract(vec3 uv, vec3 n, float etai_over_etat) {
	auto cos_theta = dot(-uv, n);
	vec3 r_out_parallel = etai_over_etat * (uv + cos_theta * n);
	vec3 r_out_perp = -sqrt(1.0f - (length(r_out_parallel)) * (r_out_parallel)) * n;
	return r_out_parallel + r_out_perp;
}
__device__ vec3 ofset(ray r, vec3 origin) {
	return origin;
}
///@param r The ray with which the the intersection is done.
///@param sky A pointer to the sky texture character array.
///@param su The u co-ordinate of the sky pixel which needs to be projected.
///@param sv the v co-ordinate of the sky pixel which needs to be projected.
///@param sc The device pointer to the memory segment where the scene was set up. 
///@param local_rand_state pointer to the CuRand state associated with current pixel.
///@param depth The maximum recursion depth.
///
///Returns the pixel color by recursively multiplying the attenuated colors, after 10 light bounces.
/// The __device__ declspec means that this function can only be called on the GPU.
__device__ vec3 pix_data3(ray r, unsigned char* sky, int su, int sv, scene** sc, hiprandState* local_rand_state, int depth) {
	if (depth <= 0)
		return vec3(0.f, 0.f, 0.f);
	sphere_hit_details rec;
	bool hit = (*sc)->hit_full(r, rec);

	if (hit)
	{
		vec3 N = vec3(rec.normal.x, rec.normal.y, rec.normal.z);
		vec3 target = rec.p + rec.normal + random_in_unit_sphere(local_rand_state);
		vec3 bounce_ray_dir;
		vec3 bounce_ray_origin;
		/*if (rec.type == REFLECTIVE) {
			vec3 reflected_ray_dir = glm::normalize(reflect(r.get_direction(), N));
			bounce_ray_dir = reflected_ray_dir;
			bounce_ray_origin = rec.p;
		}*/
		if (rec.type == REFRACTIVE) {
			vec3 refracted_ray_dir = glm::normalize(refract(normalize(r.get_direction()), N, rec.ref_ind));
			bounce_ray_dir = refracted_ray_dir;
			bounce_ray_origin = rec.p;
		}
		if (rec.type == DIFFUSE) {
			vec3 target = rec.p + rec.normal + random_in_unit_sphere(local_rand_state);
			bounce_ray_dir = target - rec.p;
			bounce_ray_origin = rec.p;
		}

		ray scattered = ray(bounce_ray_origin, bounce_ray_dir);
		vec3 albedo = rec.albedo;
		/*if (dot(scattered.get_direction(), rec.normal) > 0) {
			return albedo * pix_data3(scattered, sky, su, sv, sc, local_rand_state, depth - 1);
		}*/
		return albedo * pix_data3(scattered, sky, su, sv, sc, local_rand_state, depth - 1);

		return vec3(0.f, 0.f, 0.f);
		/*return 0.5f * vec3(N.x + 1, N.y + 1, N.z + 1);*/
		//return vec3(0.f, 0.f, 0.5f) * pix_data3(ray(rec.p, target - rec.p), sky, su, sv, sc, local_rand_state, depth -1);
	}

	else
	{
		/*vec3 sky_col;
		int index = sv * 1920 * 3 + su * 3;
		int r = (int)sky[index];
		float rc = (float)((float)r / 255);
		int g = (int)sky[index + 1];
		float gc = (float)((float)g / 255);
		int b = (int)sky[index + 2];
		float bc = (float)((float)b / 255);
		sky_col.x = rc;
		sky_col.y = gc;
		sky_col.z = bc;
		return sky_col;*/
		vec3 unit_direction = glm::normalize(r.get_direction());
		float t = 0.5f * (unit_direction.y + 1.0f);
		return (1.0f - t) * vec3(1.0f, 1.0f, 1.0f) + t * vec3(0.9, 0.7, 1.0);
	}
}
///@param pix_buff_loc A device pointer to the segment of memory on the GPU with the OpenGL pixel buffer object.
///@param sky A pointer to the sky texture character array.
///@param max_x The index of the left_most pixel of the frame buffer.
///@param max_y The index of the right most pixel of the frame buffer.
///@param rand_state pointer to the array of CuRand states.
///@param sc Device pointer to the segment of memory containing the scene object.
///
///Populates the pixel buffer with appropriate pixel colors and intensities, by implementing the ray tracing algorithm. MultiSample AntiAliasing is also implemented(Sample Count = 100).
/// The __device__ declspec means that this function can only be called on the GPU.
__global__ void render(unsigned char* pix_buff_loc, int max_x, int max_y, unsigned char* sky, scene** sc, hiprandState* rand_state) {
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	if ((i >= max_x) || (j >= max_y)) return;
	int pixel_index = j * max_x * 4 + i * 4;
	hiprandState local_rand_state = rand_state[(int)pixel_index / 100];
	camera c;
	vec3 col(0, 0, 0);
	float sample_count = 100.f;
	for (int s = 0; s < sample_count; s++) {
		float u = float(i + hiprand_uniform(&local_rand_state)) / float(max_x);
		float v = float(j + hiprand_uniform(&local_rand_state)) / float(max_y);
		ray r1 = c.get_ray(u, v);

		col += pix_data3(r1, sky, i, j, sc, &local_rand_state, 10);
	}
	col = col / sample_count;

	unsigned char r = (int)(255 * col.x);
	unsigned char g = (int)(255 * col.y);
	unsigned char b = (int)(255 * col.z);
	//remove the r+1 when not using the texture
	pix_buff_loc[pixel_index + 0] = (int)r + 1;
	pix_buff_loc[pixel_index + 1] = (int)g;
	pix_buff_loc[pixel_index + 2] = (int)b;
	pix_buff_loc[pixel_index + 3] = 255;
}
///@param rand_state Pointer to the GPU memory segment where the CuRand states are to be initailized.
///
///Initializes CUDA random states at the given device pointer.
__global__ void render_init(hiprandState* rand_state) {
	int index = blockDim.x + threadIdx.x;
	hiprand_init(1984, index, 0, &rand_state[index]);
}
///@param sph Device pointer to thge GPU memory segment where the spheres are to be initialized.
///@param count The number of spheres in the scene.
///
///Initializes the sphere objects, modify body to add remove, or resize the spheres.
__global__ void add_spheres(sphere** sph, int count) {
	*(sph) = new  sphere(vec3(-.5f, .00005f, -2.5f), .5f, vec3(1.f, 1.f, 1.f), REFRACTIVE, 1.7f);
	*(sph + 1) = new sphere(vec3(.5f, .00005f, -1.5f), .35f, vec3(0.9f, 0.1f, 0.98f), DIFFUSE, 0.f);
	*(sph + 2) = new sphere(vec3(0.f, -100.5f, -1.f), 100.f, vec3(0.15f, 0.996f, 0.15f), DIFFUSE, 0.f);
	*(sph + 3) = new sphere(vec3(1.5f, .00005f, -2.5f), .5f, vec3(0.98f, 0.2f, 0.2f), DIFFUSE, 0.f);
	*(sph + 4) = new sphere(vec3(-1.5f, .00005f, -2.5f), .5f, vec3(0.2f, 0.2f, 0.992f), DIFFUSE, 0.f);
}
///The main driver, sets up the GLFW OpenGL context, then the Textures and Pixel Buffer Object. Also renders the frame buffer to the Context. The Device pointers, shared memory, OpenGL interop are al configured within the body. The thread dispatch is also configured.
/// Effectively the mfunction draws a fullscreen quad onto the context and then texture maps the ray traced pixel buffer which is shared with CUDA onto the Quad.
int main()
{
	hipSetDevice(0);

	GLFWwindow* window;
	if (!glfwInit())
		return -1;
	window = glfwCreateWindow(1920, 1080, "CUDA project", glfwGetPrimaryMonitor(), NULL);
	if (!window)
	{
		glfwTerminate();
		return -1;
	}
	glfwMakeContextCurrent(window);
	glewInit();
	float vertices[] = {
		// positions          // colors           // texture coords
		1.f,  1.f, 0.0f,   1.0f, 0.0f, 0.0f,   1.0f, 1.0f, // top right
		1.f, -1.f, 0.0f,   0.0f, 1.0f, 0.0f,   1.0f, 0.0f, // bottom right
		-1.f, -1.f, 0.0f,   0.0f, 0.0f, 1.0f,   0.0f, 0.0f, // bottom left
		-1.f,  1.f, 0.0f,   1.0f, 1.0f, 0.0f,   0.0f, 1.0f  // top left
	};
	unsigned int indices[] = {
		0, 1, 3,
		1, 2, 3
	};

	VertexBuffer vb(vertices, sizeof(vertices));
	IndexBuffer eb(indices, 6);
	VertexArray va;

	va.spec_vertex_size(8);
	va.add_layout_spec(3);
	va.add_layout_spec(3);
	va.add_layout_spec(2);
	va.AddBuffer(vb);
	Texture t;
	int width, height, nrChannels;
	width = 1920;
	height = 1080;
	nrChannels = 4;

	unsigned int pbo;
	glGenBuffers(1, &pbo);
	glBindBuffer(GL_PIXEL_UNPACK_BUFFER, pbo);
	glBufferData(GL_PIXEL_UNPACK_BUFFER, width * height * nrChannels * sizeof(GLubyte), NULL, GL_DYNAMIC_DRAW);
	glBindBuffer(GL_PIXEL_UNPACK_BUFFER, 0);
	hipGraphicsResource* res;
	gpuCheckErrs(hipGraphicsGLRegisterBuffer(&res, pbo, cudaGraphicsMapFlagsNone));
	gpuCheckErrs(hipGraphicsMapResources(1, &res, 0));
	unsigned char* out_data;
	size_t num_bytes;
	gpuCheckErrs(hipGraphicsResourceGetMappedPointer((void**)&out_data, &num_bytes, res));

	int tx = 8;//threads x
	int ty = 8;//threads y
	dim3 blocks(width / tx + 1, height / ty + 1);
	dim3 threads(tx, ty);

	hiprandState* d_rand_state;
	gpuCheckErrs(hipMalloc((void**)&d_rand_state, sizeof(hiprandState)));
	render_init << <512, 108 >> > (d_rand_state);

	//setting up the sky

	int w, h, n;
	stbi_set_flip_vertically_on_load(true);
	unsigned char* data = stbi_load("res/textures/sky3.jpg", &w, &h, &n, 0);
	unsigned char* sky;
	hipMalloc(&sky, w * h * 3);
	hipMemcpy(sky, data, w * h * 3, hipMemcpyHostToDevice);

	//setting up the rest of the scene

	sphere** spheres;
	hipMalloc(&spheres, sizeof(sphere*) * 5);
	add_spheres << < 1, 1 >> > (spheres, 5);

	scene** sc;
	hipMalloc(&sc, sizeof(scene*));
	make_scene << < 1, 1 >> > (spheres, sc, 5);

	vec3 lower_left_corner(-1.6, -0.9, -1.0);
	vec3 horizontal(3.2, 0.0, 0.0);
	vec3 vertical(0.0, 1.8, 0.0);
	vec3 origin(0.0, 0.0, 0.0);
	render << <blocks, threads >> > (out_data, width, height, sky, sc, d_rand_state);
	hipGraphicsUnmapResources(1, &res);

	glBindBuffer(GL_PIXEL_UNPACK_BUFFER, pbo);
	glUnmapBuffer(GL_PIXEL_UNPACK_BUFFER);
	t.use_pbo(width, height);
	glBindBuffer(GL_PIXEL_UNPACK_BUFFER, 0);

	Shader s("res/shaders/tex_basic.shader");
	glfwSetCursorPosCallback(window, MouseControlWrapper);
	glfwSetScrollCallback(window, ScrollControlWrapper);
	glEnable(GL_DEPTH_TEST);
	glEnable(GL_BLEND);
	glBlendFunc(GL_SRC_ALPHA, GL_ONE_MINUS_SRC_ALPHA);
	glDisable(GL_CULL_FACE);

	while (!glfwWindowShouldClose(window))
	{
		primary_cam.input_handler(window);
		glClearColor(0.f, 0.f, 0.f, 1.0f);
		glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
		s.Bind();
		t.Bind();
		va.Bind();
		eb.Bind();
		glDrawElements(GL_TRIANGLES, 6, GL_UNSIGNED_INT, 0);
		va.Unbind();
		s.Unbind();
		/* Swap front and back buffers */
		glfwSwapBuffers(window);

		/* Poll for and process events */
		glfwPollEvents();
	}
	glfwTerminate();
	return 0;
}