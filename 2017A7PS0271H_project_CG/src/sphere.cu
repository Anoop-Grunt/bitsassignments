#include "hip/hip_runtime.h"
#include "sphere.cuh"


sphere::sphere(vec3 center, float r, vec3 albedo, mat_type m, float ref_index)
    :center(center), radius(r), albedo(albedo), type(m), ri(ref_index)
{
}

sphere::~sphere()
{
}

__device__ bool sphere::hit(ray r, float t_min, float t_max, sphere_hit_details& record) {

    vec3 oc = r.get_origin() - center;
    float a = dot(r.get_direction(), r.get_direction());
    float h = dot(oc, r.get_direction());
    float c = dot(oc, oc) - radius * radius;
    float discriminant = h * h - a * c;
    if (discriminant > -0.f){

        // if both roots are real and unequal, the ray hits at two points, in which case there will be an incoming and outgoing ray
        //For the incoming ray we set  the normal in the same direction as (r(t) -cen)
        //but for outgoing ray we flip the normal, so that it points outwards from the sphere
        //the tangential case is obvioslu pretty rare, but in tht case the discrimnant is zeo
        float temp = (-h - sqrt(discriminant)) / a;
        if (temp < t_max && temp > t_min) {
            record.albedo = albedo;
            record.t = temp;
            record.p = r.get_point_at_t(record.t);
            record.normal = (record.p - center) / radius;
            vec3 outward_normal = (record.p - center) / radius;
            record.orient_normal(r, outward_normal);
            if (type == REFLECTIVE) {
                record.type = REFLECTIVE;
            }
            if (type == REFRACTIVE) {
                record.type = REFRACTIVE;
                if (record.front_face) {
                    record.ref_ind = 1 / ri;
                    
                }
                else
                {
                    record.ref_ind = ri;
                }

            }
            if (type == DIFFUSE) {
                record.type = DIFFUSE;
                
            }
            
            return true;

        }

        temp = (-h + sqrt(discriminant)) / a;

        if (temp < t_max && temp > t_min) {
            record.albedo = albedo;
            record.t = temp;
            record.p = r.get_point_at_t(record.t);
            record.normal = (record.p - center) / radius;
            vec3 outward_normal = (record.p - center) / radius;
            record.orient_normal(r, outward_normal);
            if (type == REFLECTIVE) {
                record.type = REFLECTIVE;
            }
            if (type == REFRACTIVE) {
                record.type = REFRACTIVE;
                if (record.front_face) {
                    record.ref_ind = 1 / ri;

                }
                else
                {
                    record.ref_ind = ri;
                }
            }
            if (type == DIFFUSE) {
                record.type = DIFFUSE;
            }

            return true;
        }

    }

    return false;
}
